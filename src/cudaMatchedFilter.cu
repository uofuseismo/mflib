
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipfft/hipfft.h>

typedef float2  CComplex;
typedef double2 ZComplex;
template<typename T>
static __device__ __host__ inline T operator+(const T a, const T b);
template<typename T>
static __device__ __host__ inline T operator*(const T a, const T b);

// Complex addition
template<typename T> static __device__ __host__ 
inline T operator+(const T a, const T b)
{
    T c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}
// Complex multiplication
template<typename T> static __device__ __host__
inline T operator*(const T a, const T b)
{
    T c;
    c.x = a.x*b.x - a.y*b.y;
    c.y = a.x*b.y + a.y*b.x;
    return c;
}

/// @brief Computes: \f$ X \leftarrow B X  \f$.
/// @param[in,out] x    On input this is the spectra of the waveform.
///                     On exit, this is the spectra multiplied with the
///                     spectra of the filter - i.e., the convolution.
///                     This is an array whose dimension is [nw].
/// @param[in] b        The spectra of the filter coefficients.
///                     This is an array whose dimension is [nw].
/// @param[in] nw       The number of frequencies. 
static __global__
void multiplySpectra(CComplex *x, const CComplex *b, const int nw)
{
    const int numThreads = blockDim.x*gridDim.x;
    const int threadID   = blockIdx.x*blockDim.x + threadIdx.x;
    for (int i=threadID; i<nw; i=i+numThreads)
    {
        x[i] = x[i]*b[i];
    }
}


struct cufft32z_struct
{
    hipfftHandle mPlan;
};

extern "C"
void clear(struct cufft32z_struct *cuft);

extern "C"
void initialize(struct cufft32z_struct *cuft);


void clear(struct cufft32z_struct *cuft)
{
    hipfftDestroy(cuft->mPlan);
}

void initialize(struct cufft32z_struct *cuft)
{
    //checkCudaErrors(cufftMakePlan1d(plan_input, new_size, CUFFT_C2C, 1, worksize));  
}

